#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include "gpumat.h"
#include "cuda_common.h"
#include "common_types.h"

#include "common_devices.h"
#include "cuda_types.h"

using namespace gpumat;

///////// begin internal namespace ///////////////

namespace gpumat{

namespace internal{

template< typename T >
__device__ void _im2cols(const Mtx& X, const ct::Size& szA0, int channels, const ct::Size& szW, int stride, Mtx& Res, const ct::Size& szOut)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szOutArea = szOut.width * szOut.height;
	int all = szOutArea * channels;

	if(col < all){
		int c = col / szOutArea;
		int offset = col - c * szOutArea;

		int y = offset / szOut.width;
		int x = offset - y * szOut.width;

		int x0 = x * stride;
		int y0 = y * stride;
		int row2 = y * szOut.width + x;

		int szA0area = szA0.width * szA0.height;
		int szWarea = szW.width * szW.height;

		T *dX = (T*)X.data;
		T *dR = (T*)Res.data;
		T *dXi = &dX[c * szA0area];

		for(int a = 0; a < szW.height; ++a){

			for(int b = 0; b < szW.width; ++b){

				int col2 = c * szWarea + (a * szW.width + b);
				T val = 0;
				if(y0 + a < szA0.height && x0 + b < szA0.width){
					val = dXi[(y0 + a) * szA0.width + (x0 + b)];
				}
				if(col2 < Res.cols)
					dR[row2 * Res.cols + col2] = val;
			}
		}
	}
}

template< typename T >
__device__ void _im2colsT(const Mtx& X, const ct::Size& szA0, int channels, const ct::Size& szW, int stride, Mtx& Res, const ct::Size& szOut)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szOutArea = szOut.width * szOut.height;
	int all = szOutArea * channels;

	if(col < all){
		int c = col / szOutArea;
		int offset = col - c * szOutArea;

		int y = offset / szOut.width;
		int x = offset - y * szOut.width;

		int x0 = x * stride;
		int y0 = y * stride;
		int row2 = y * szOut.width + x;

//		int szA0area = szA0.width * szA0.height;
		int szWarea = szW.width * szW.height;

		T *dR = (T*)Res.data;
		T *dXi = (T*)X.data + c;

		for(int a = 0; a < szW.height; ++a){
			
			for(int b = 0; b < szW.width; ++b){
				
				int col2 = c * szWarea + (a * szW.width + b);
				T val = 0;
				if(y0 + a < szA0.height && x0 + b < szA0.width){
					val = dXi[((y0 + a) * szA0.width + (x0 + b)) * channels];
				}
				if(col2 < Res.cols)
					dR[row2 * Res.cols + col2] = val;
			}
		}
	}
}

template< typename T >
__global__ void im2cols(Mtx X, ct::Size szA0, int channels, ct::Size szW, int stride, Mtx Res, ct::Size szOut)
{
	_im2cols<T>(X, szA0, channels, szW, stride, Res, szOut);
}

template< typename T >
__global__ void im2cols_vec(SmallMtxArray X, ct::Size szA0, int channels, ct::Size szW, int stride, SmallMtxArray Res, ct::Size szOut)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_im2cols<T>(X.mtx[row], szA0, channels, szW, stride, Res.mtx[row], szOut);
	}
}

////////

template< typename T >
__global__ void im2colsT(Mtx X, ct::Size szA0, int channels, ct::Size szW, int stride, Mtx Res, ct::Size szOut)
{
	_im2colsT<T>(X, szA0, channels, szW, stride, Res, szOut);
}

template< typename T >
__global__ void im2colsT_vec(SmallMtxArray X, ct::Size szA0, int channels, ct::Size szW, int stride, SmallMtxArray Res, ct::Size szOut)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_im2colsT<T>(X.mtx[row], szA0, channels, szW, stride, Res.mtx[row], szOut);
	}
}

//////// begin same //////

template< typename T >
__device__ void _im2colsSame(const Mtx& X, const ct::Size& szA0, int channels, const ct::Size& szW, int stride, Mtx& Res, const ct::Size& szOut)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szOutArea = szOut.width * szOut.height;
	int all = szOutArea * channels;

	if(col < all){
		int c = col / szOutArea;
		int offset = col - c * szOutArea;

		int y = offset / szOut.width;
		int x = offset - y * szOut.width;

		int x0 = x * stride;
		int y0 = y * stride;
		int row2 = y * szOut.width + x;

		if(row2 >= Res.rows)
			return;

		int szA0area = szA0.width * szA0.height;
		int szWarea = szW.width * szW.height;

		T *dX = (T*)X.data;
		T *dR = (T*)Res.data;
		T *dXi = &dX[c * szA0area];

		for(int _a = 0; _a < szW.height; ++_a){
			int a = _a - szW.height/2;
			for(int _b = 0; _b < szW.width; ++_b){
				int b = _b - szW.width/2;
				int col2 = c * szWarea + (_a * szW.width + _b);
				T val = 0;
				if(y0 + a >= 0 && y0 + a < szA0.height && x0 + b >= 0 && x0 + b < szA0.width){
					val = dXi[(y0 + a) * szA0.width + (x0 + b)];
				}
				if(col2 < Res.cols)
					dR[row2 * Res.cols + col2] = val;
			}
		}
	}
}

template< typename T >
__device__ void _im2colsTSame(const Mtx& X, const ct::Size& szA0, int channels, const ct::Size& szW, int stride, Mtx& Res, const ct::Size& szOut)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szOutArea = szOut.width * szOut.height;
	int all = szOutArea * channels;

	if(col < all){
		int c = col / szOutArea;
		int offset = col - c * szOutArea;

		int y = offset / szOut.width;
		int x = offset - y * szOut.width;

		int x0 = x * stride;
		int y0 = y * stride;
		int row2 = y * szOut.width + x;

		if(row2 >= Res.rows)
			return;

//		int szA0area = szA0.width * szA0.height;
		int szWarea = szW.width * szW.height;

		T *dR = (T*)Res.data;
		T *dXi = (T*)X.data + c;

		for(int _a = 0; _a < szW.height; ++_a){
            int ya = y0 + _a - szW.height/2;
            if(ya >= 0 && ya < szA0.height){
                int yaw = (ya) * szA0.width;
                for(int _b = 0; _b < szW.width; ++_b){
                    int xb = x0 + _b - szW.width/2;
                    int col2 = c * szWarea + (_a * szW.width + _b);
                    T val = 0;
                    if(xb >= 0 && xb < szA0.width){
                        val = dXi[(yaw + xb) * channels];
                    }
                    if(col2 < Res.cols)
                        dR[row2 * Res.cols + col2] = val;
                }
            }
		}
	}
}

template< typename T >
__global__ void im2colsSame(Mtx X, ct::Size szA0, int channels, ct::Size szW, int stride, Mtx Res, ct::Size szOut)
{
	_im2colsSame<T>(X, szA0, channels, szW, stride, Res, szOut);
}

template< typename T >
__global__ void im2cols_vecSame(SmallMtxArray X, ct::Size szA0, int channels, ct::Size szW, int stride, SmallMtxArray Res, ct::Size szOut)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_im2colsSame<T>(X.mtx[row], szA0, channels, szW, stride, Res.mtx[row], szOut);
	}
}

////////

template< typename T >
__global__ void im2colsTSame(Mtx X, ct::Size szA0, int channels, ct::Size szW, int stride, Mtx Res, ct::Size szOut)
{
	_im2colsTSame<T>(X, szA0, channels, szW, stride, Res, szOut);
}

template< typename T >
__global__ void im2colsT_vecSame(SmallMtxArray X, ct::Size szA0, int channels, ct::Size szW, int stride, SmallMtxArray Res, ct::Size szOut)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_im2colsTSame<T>(X.mtx[row], szA0, channels, szW, stride, Res.mtx[row], szOut);
	}
}


//////// end same ////////

template< typename T >
__device__ void _cols2im(const Mtx& Delta,
				 const ct::Size& szOut,
				 const ct::Size& szA0,
				 int channels,
				 const ct::Size& szW,
				 int stride,
				 Mtx X)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szA0Area = szA0.width * szA0.height;
	int all = szA0Area * channels;
	if(col < all){
//		int c = col / szOutArea;
//		int offset = col - c * szOutArea;

//		int y = offset / szOut.width;
//		int x = offset - y * szOut.width;

//		int x0 = x * stride;
//		int y0 = y * stride;
//		int row2 = y * szOut.width + x;

//		int szA0area = szA0.width * szA0.height;
//		int szWarea = szW.width * szW.height;

//		T *dX = (T*)X.data;
//		T *dR = (T*)Delta.data;
//		T *dXi = &dX[c * szA0area];

//		for(int a = 0; a < szW.height; ++a){
//			for(int b = 0; b < szW.width; ++b){
//				int col2 = c * szWarea + (a * szW.width + b);
//				if(y0 + a < szA0.height && x0 + b < szA0.width){
//					dXi[(y0 + a) * szA0.width + (x0 + b)] += dR[row2 * Delta.cols + col2];
//				}
//			}
//		}
		int c = col / szA0Area;
		int offset = col - c * szA0Area;

		int y = offset / szA0.width;
		int x = offset - y * szA0.width;

		int szWarea = szW.width * szW.height;

		T *dX = (T*)X.data;
		T *dR = (T*)Delta.data;
		T *dXi = &dX[c * szA0Area];

		T sum = 0;
		for(int a = 0; a < szW.height; ++a){
			if((y - a) % stride == 0){
				int y0 = (y - a) / stride;
				if(y0 >= 0 && y0 < szOut.height){
					for(int b = 0; b < szW.width; ++b){

						if((x - b) % stride == 0){

							int x0 = (x - b) / stride;

							if(x0 >= 0 && x0 < szOut.width){
								int row2 = y0 * szOut.width + x0;
								int col2 = c * szWarea + (a * szW.width + b);
								T val = dR[row2 * Delta.cols + col2];
								sum += val;
							}
						}
					}
				}
			}
		}
		dXi[y * szA0.width + x] = sum;

	}
}

//////

template< typename T >
__device__ void _cols2imT(const Mtx& Delta,
				 const ct::Size& szOut,
				 const ct::Size& szA0,
				 int channels,
				 const ct::Size& szW,
				 int stride,
				 Mtx X)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szA0Area = szA0.width * szA0.height;
	int all = szA0Area * channels;

	if(col < all){
		int c = col / szA0Area;
		int offset = col - c * szA0Area;

		int y = offset / szA0.width;
		int x = offset - y * szA0.width;

//		int x0 = x * stride;
//		int y0 = y * stride;
//		int row2 = y * szOut.width + x;

//		int szA0area = szA0.width * szA0.height;
		int szWarea = szW.width * szW.height;

		T *dR = (T*)Delta.data;
		T *dXi = (T*)X.data + c;

//		for(int a = 0; a < szW.height; ++a){
//			for(int b = 0; b < szW.width; ++b){
//				int col2 = c * szWarea + (a * szW.width + b);
//				if(y0 + a < szA0.height && x0 + b < szA0.width){
//					dXi[((y0 + a) * szA0.width + (x0 + b)) * channels] += dR[row2 * Delta.cols + col2];
//				}
//			}
//		}
		T sum = 0;
		for(int a = 0; a < szW.height; ++a){
			if((y - a) % stride == 0){
				int y0 = (y - a) / stride;
				if(y0 >= 0 && y0 < szOut.height){
					for(int b = 0; b < szW.width; ++b){

						if((x - b) % stride == 0){

							int x0 = (x - b) / stride;

							if(x0 >= 0 && x0 < szOut.width){
								int row2 = y0 * szOut.width + x0;
								int col2 = c * szWarea + (a * szW.width + b);
								T val = dR[row2 * Delta.cols + col2];
								sum += val;
							}
						}
					}
				}
			}
		}
		dXi[(y * szA0.width + x) * channels] = sum;
	}
}

/////

template< typename T >
__global__ void cols2im(Mtx Delta,
						   ct::Size szOut,
						   ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   Mtx X)
{
	_cols2im<T>(Delta, szOut, szA0, channels, szW, stride, X);
}

template< typename T >
__global__ void cols2im_vec(SmallMtxArray Delta,
						   ct::Size szOut,
						   ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   SmallMtxArray X)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_cols2im<T>(Delta.mtx[row], szOut, szA0, channels, szW, stride, X.mtx[row]);
	}
}

////////

template< typename T >
__global__ void cols2imT(Mtx Delta,
						   ct::Size szOut,
						   ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   Mtx X)
{
	_cols2imT<T>(Delta, szOut, szA0, channels, szW, stride, X);
}

template< typename T >
__global__ void cols2imT_vec(SmallMtxArray Delta,
						   ct::Size szOut,
						   ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   SmallMtxArray X)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_cols2imT<T>(Delta.mtx[row], szOut, szA0, channels, szW, stride, X.mtx[row]);
	}
}

//////////////// SAME ///////////

template< typename T >
__device__ void _cols2im_same(const Mtx& Delta,
				 ct::Size szDelta, const ct::Size& szA0,
				 int channels,
				 const ct::Size& szW,
				 int stride,
				 Mtx X)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szA0Area = szA0.width * szA0.height;
	int all = szA0Area * channels;
	if(col < all){
		int c = col / szA0Area;
		int offset = col - c * szA0Area;

		int y = offset / szA0.width;
		int x = offset - y * szA0.width;

		int szWarea = szW.width * szW.height;

		T *dX = (T*)X.data;
		T *dR = (T*)Delta.data;
		T *dXi = &dX[c * szA0Area];

		T sum = 0;
		for(int _a = 0; _a < szW.height; ++_a){
			int a = _a - szW.height / 2;
			if((y - a) % stride == 0){
				int y0 = (y - a) / stride;
				if(y0 >= 0 && y0 < szDelta.height){
					for(int _b = 0; _b < szW.width; ++_b){

						int b = _b - szW.width/2;
						if((x - b) % stride == 0){

							int x0 = (x - b) / stride;
							T val = 0;
							if(x0 >= 0 && x0 < szDelta.width){
								int row2 = y0 * szDelta.width + x0;
								int col2 = c * szWarea + (_a * szW.width + _b);
								if(row2 < Delta.rows && col2 < Delta.cols)
									val = dR[row2 * Delta.cols + col2];
								sum += val;
							}
						}
					}
				}
			}
		}
		dXi[y * szA0.width + x] = sum;

	}
}

template< typename T >
__device__ void _cols2imT_same(const Mtx& Delta,
				 ct::Size szDelta, const ct::Size& szA0,
				 int channels,
				 const ct::Size& szW,
				 int stride,
				 Mtx X)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szA0Area = szA0.width * szA0.height;
	int all = szA0Area * channels;

	if(col < all){
		int c = col / szA0Area;
		int offset = col - c * szA0Area;

		int y = offset / szA0.width;
		int x = offset - y * szA0.width;

//		int x0 = x * stride;
//		int y0 = y * stride;
//		int row2 = y * szOut.width + x;

//		int szA0area = szA0.width * szA0.height;
		int szWarea = szW.width * szW.height;

		T *dR = (T*)Delta.data;
		T *dXi = (T*)X.data + c;

//		for(int a = 0; a < szW.height; ++a){
//			for(int b = 0; b < szW.width; ++b){
//				int col2 = c * szWarea + (a * szW.width + b);
//				if(y0 + a < szA0.height && x0 + b < szA0.width){
//					dXi[((y0 + a) * szA0.width + (x0 + b)) * channels] += dR[row2 * Delta.cols + col2];
//				}
//			}
//		}
		T sum = 0;
		for(int _a = 0; _a < szW.height; ++_a){
			int a = _a - szW.height/2;
			if((y - a) % stride == 0){
				int y0 = (y - a) / stride;
				if(y0 >= 0 && y0 < szDelta.height){
					for(int _b = 0; _b < szW.width; ++_b){

						int b = _b - szW.width/2;
						if((x - b) % stride == 0){

							int x0 = (x - b) / stride;
							T val = 0;
							if(x0 >= 0 && x0 < szDelta.width){
								int row2 = y0 * szDelta.width + x0;
								int col2 = c * szWarea + (_a * szW.width + _b);
								if(row2 < Delta.rows && col2 < Delta.cols)
									val = dR[row2 * Delta.cols + col2];
								sum += val;
							}
						}
					}
				}
			}
		}
		dXi[(y * szA0.width + x) * channels] = sum;
	}
}

template< typename T >
__global__ void cols2im_same(Mtx Delta,
						   ct::Size szDelta, ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   Mtx X)
{
	_cols2im_same<T>(Delta, szDelta, szA0, channels, szW, stride, X);
}

template< typename T >
__global__ void cols2im_vec_same(SmallMtxArray Delta,
						   ct::Size szDelta, ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   SmallMtxArray X)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_cols2im_same<T>(Delta.mtx[row], szDelta, szA0, channels, szW, stride, X.mtx[row]);
	}
}

//////

template< typename T >
__global__ void cols2imT_same(Mtx Delta,
						   ct::Size szDelta, ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   Mtx X)
{
	_cols2imT_same<T>(Delta, szDelta, szA0, channels, szW, stride, X);
}

template< typename T >
__global__ void cols2imT_vec_same(SmallMtxArray Delta,
						   ct::Size szDelta, ct::Size szA0,
						   int channels,
						   ct::Size szW,
						   int stride,
						   SmallMtxArray X)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_cols2imT_same<T>(Delta.mtx[row], szDelta, szA0, channels, szW, stride, X.mtx[row]);
	}
}

/////////////////////////////////

template< typename T>
__device__ T min(T a, T b)
{
    return a < b? a : b;
}

template< typename T >
__device__ void _subsample(const Mtx &X,
						   int K,
						   const ct::Size& szA,
						   Mtx Y,
						   Mtx Mask,
						   const ct::Size& szO)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szOutArea = szO.width * szO.height;
	int all = szOutArea * K;

	const int stride = 2;

	if(col < all){
		int k = col / szOutArea;
		int offset = col - k * szOutArea;

		int y = offset / szO.width;
		int x = offset - y * szO.width;

		T *dX = (T*)X.data + k;
		T* dM = (T*)Mask.data + k;
		T *dY = (T*)Y.data + k;

		int y0 = y * stride;
		int x0 = x * stride;

		T mmax = dX[(y0 * szA.width + x0) * X.cols];
		int xm = x0, ym = y0;

        int yr = min(y0 + stride, szA.height);
        int xr = min(x0 + stride, szA.width);
        for(int ya = y0; ya < yr; ++ya){
            int y0a = (ya) * szA.width;
            for(int xb = x0; xb < xr; ++xb){
                T val = dX[(y0a + xb) * X.cols];
                if(val > mmax){
                    mmax = val;
                    xm = xb;
                    ym = ya;
                }
            }
		}

		dY[(y * szO.width + x) * Y.cols] = mmax;
		dM[(ym * szA.width + xm) * Mask.cols] = 1;
	}
}

template< typename T >
__global__ void subsample(Mtx X,
						  int K,
						  ct::Size szA,
						  Mtx Y,
						  Mtx Mask,
						  ct::Size szO)
{
	_subsample<T>(X, K, szA, Y, Mask, szO);
}

template< typename T >
__global__ void subsample_vec(SmallMtxArray X,
						  int K,
						  ct::Size szA,
						  SmallMtxArray Y,
						  SmallMtxArray Mask,
						  ct::Size szO)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_subsample<T>(X.mtx[row], K, szA, Y.mtx[row], Mask.mtx[row], szO);
	}
}

template< typename T >
__device__ void _upsample(const Mtx &Y,
						 const Mtx &Mask,
						 int K,
						 const ct::Size &szO,
						 const ct::Size &szA,
						 Mtx X)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int szOutArea = szO.width * szO.height;
	int all = szOutArea * K;

	int stride = 2;

	if(col < all){
		int k = col / szOutArea;
		int offset = col - k * szOutArea;

		int y = offset / szO.width;
		int x = offset - y * szO.width;

		T *dX = (T*)(X.data) + k;
		T* dM = (T*)(Mask.data) + k;
		T *dY = (T*)(Y.data) + k;

		int y0 = y * stride;
		int x0 = x * stride;

		T val = dY[(y * szO.width + x) * K];

        int yr = min(y0 + stride, szA.height);
        int xr = min(x0 + stride, szA.width);
        for(int ya = y0; ya < yr; ++ya){
            int y0a = (ya) * szA.width;
            for(int xb = x0; xb < xr; ++xb){
                T m = dM[(y0a + (xb)) * Mask.cols];
                dX[(y0a + (xb)) * X.cols] = val * m;
            }
		}
	}
}

template< typename T >
__global__ void upsample(Mtx Y,
						 Mtx Mask,
						 int K,
						 ct::Size szO,
						 ct::Size szA,
						 Mtx X)
{
	_upsample<T>(Y, Mask, K, szO, szA, X);
}

template< typename T >
__global__ void upsample_vec(SmallMtxArray Y,
							 SmallMtxArray Mask,
							 int K,
							 ct::Size szO,
							 ct::Size szA,
							 SmallMtxArray X)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < X.count){
		_upsample<T>(Y.mtx[row], Mask.mtx[row], K, szO, szA, X.mtx[row]);
	}
}

template< typename T >
__global__ void vec2mat(SmallMtxArray vec, Mtx mat)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < mat.rows && col < mat.cols){
		T* dV = (T*)vec.mtx[row].data;
		T* dM = (T*)mat.data;

		dM[row * mat.cols + col] = dV[col];
	}
}

template< typename T >
__global__ void mat2vec(Mtx mat, ct::Size sz, SmallMtxArray vec)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < mat.rows && col < mat.cols){
		T* dV = (T*)vec.mtx[row].data;
		T* dM = (T*)mat.data;

		int y = col/sz.width;
		int x = col - y * sz.width;

		dV[y * sz.width + x] = dM[row * mat.cols + col];
	}
}

template< typename T >
__global__ void addvec(Mtx mat,  SmallMtxArray vec, T alpha)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if(row < mat.rows && col < mat.cols){
		T* dM = (T*)mat.data;

		T val = 0;
		for(int i = 0; i < vec.count; ++i){
			T* dV = (T*)vec.mtx[i].data;
			val += dV[row * mat.cols + col];
		}

		dM[row * mat.cols + col] = val * alpha;
	}
}

__device__ void get_offset_and_cnt(int& off, int& cnt, int count, int rowi)
{
    off = rowi * BLOCKSIZE_CNV;
	cnt = count - off;
    cnt = max(0, min(BLOCKSIZE_CNV, cnt));
}

template< typename T >
__device__ T sqr(T val)
{
	return val * val;
}

template< typename T >
__global__ void get_mean(const SmallMtxArray X,  Mtx Mean, int spatial, int channels)
{
//	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	T N = X.count * spatial, val;
	int offr, cntr;
	int coli = threadIdx.x; int rowi = threadIdx.y;
	get_offset_and_cnt(offr, cntr, X.count, rowi);

    __shared__ T data[BLOCKSIZE_CNV][BLOCKSIZE_CNV];

	if(col < channels){

		T *dMean	= (T*)Mean.data;

		val = 0;
		for(int i = 0; i < cntr; ++i){
			T *dX = (T*)X.mtx[offr + i].data;
			for(int s = 0; s < spatial; ++s) val += dX[col + s * channels];
		}
		data[rowi][coli] = val;

		__syncthreads();

		if(rowi == 0){
			val = 0;
			for(int i = 0; i < cntr; ++i) val += data[i][coli];
			dMean[col] = val / N;
		}
	}
}

template< typename T >
__global__ void get_var(const SmallMtxArray X, const Mtx Mean, SmallMtxArray Xmu, Mtx Var, int spatial, int channels)
{
	T eps = 1e-8;
//	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	T N = X.count * spatial, val;
	int offr, cntr;
	int coli = threadIdx.x; int rowi = threadIdx.y;
	get_offset_and_cnt(offr, cntr, X.count, rowi);

    __shared__ T data[BLOCKSIZE_CNV][BLOCKSIZE_CNV];

	if(col < channels){

		T *dMean	= (T*)Mean.data;
		T *dV		= (T*)Var.data;

		val = 0;
		for(int i = 0; i < cntr; ++i){
			T *dX = (T*)X.mtx[offr + i].data;
			T *dXmu = (T*)Xmu.mtx[offr + i].data;
			for(int s = 0; s < spatial; ++s){
				T w = dX[col + s * channels] - dMean[col];
				val += w * w;
				dXmu[col + s * channels] = w;
			}
		}
		data[rowi][coli] = val;

		__syncthreads();

		if(rowi == 0){
			val = 0;
			for(int i = 0; i < cntr; ++i) val += data[i][coli];
			dV[col] = sqrt((val / (N - 1)) + eps);
		}
	}
}

template< typename T >
__global__ void get_shift(const SmallMtxArray Xmu, const Mtx gamma, const Mtx betha,
						   const Mtx Var, SmallMtxArray Y, int spatial, int channels)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if(row < Xmu.count && col < channels){
		T *dG	= (T*)gamma.data;
		T *dB	= (T*)betha.data;
		T *dXmu		= (T*)Xmu.mtx[row].data;
		T *dVar		= (T*)Var.data;
		T *dY		= (T*)Y.mtx[row].data;

		for(int s = 0; s < spatial; ++s){
			T val = dXmu[col + s * channels] / dVar[col];
			dY[col + s * channels] = dG[col] * val + dB[col];
		}
	}
}


template< typename T >
__global__ void batch_normalize(SmallMtxArray X, Mtx Mean, Mtx Var, SmallMtxArray Xu,
								SmallMtxArray Y, Mtx alpha, Mtx betha)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if(row < X.count && col < X.mtx[0].total()){
		T *dXi		= (T*)X.mtx[row].data;
		T *dYi		= (T*)Y.mtx[row].data;
		T *dMean	= (T*)Mean.data;
		T *dAlpha	= (T*)alpha.data;
		T *dBetha	= (T*)betha.data;
		T *dXu		= (T*)Xu.mtx[row].data;
		T *dVar		= (T*)Var.data;

		T xu = dXi[col] - dMean[col];

		T val = xu / dVar[col];
		dYi[col] = dAlpha[col] * val + dBetha[col];
		dXu[col] = xu;
	}
}

}	/// @endnamespace internal

}	/// @endnamespace gpumat

extern "C"
void cuda_im2cols(const gpumat::GpuMat &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  gpumat::GpuMat &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::im2cols<double> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
		case GPU_FLOAT:
			internal::im2cols<float> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
	}
}

extern "C"
void cuda_im2cols_vec(const std::vector< gpumat::GpuMat > &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  std::vector< gpumat::GpuMat > &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	internal::SmallMtxArray sX(X), sRes(Res);

	switch (X[0].type) {
		case GPU_DOUBLE:
			internal::im2cols_vec<double> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
		case GPU_FLOAT:
			internal::im2cols_vec<float> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
	}
}

//////////

extern "C"
void cuda_im2colsT(const gpumat::GpuMat &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  gpumat::GpuMat &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::im2colsT<double> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
		case GPU_FLOAT:
			internal::im2colsT<float> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
	}
}

extern "C"
void cuda_im2colsT_vec(const std::vector< gpumat::GpuMat > &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  std::vector< gpumat::GpuMat > &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	internal::SmallMtxArray sX(X), sRes(Res);

	switch (X[0].type) {
		case GPU_DOUBLE:
			internal::im2colsT_vec<double> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
		case GPU_FLOAT:
			internal::im2colsT_vec<float> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
	}
}

////////// same

extern "C"
void cuda_im2colsSame(const gpumat::GpuMat &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  gpumat::GpuMat &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::im2colsSame<double> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
		case GPU_FLOAT:
			internal::im2colsSame<float> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
	}
}

extern "C"
void cuda_im2cols_vecSame(const std::vector< gpumat::GpuMat > &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  std::vector< gpumat::GpuMat > &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	internal::SmallMtxArray sX(X), sRes(Res);

	switch (X[0].type) {
		case GPU_DOUBLE:
			internal::im2cols_vecSame<double> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
		case GPU_FLOAT:
			internal::im2cols_vecSame<float> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
	}
}

//////////

extern "C"
void cuda_im2colsTSame(const gpumat::GpuMat &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  gpumat::GpuMat &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::im2colsTSame<double> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
		case GPU_FLOAT:
			internal::im2colsTSame<float> <<<dimGrid, dimBlock>>>(X, szA0, channels, szW, stride, Res, szOut);
			break;
	}
}

extern "C"
void cuda_im2colsT_vecSame(const std::vector< gpumat::GpuMat > &X,
				  const ct::Size &szA0,
				  int channels,
				  const ct::Size &szW,
				  int stride,
				  std::vector< gpumat::GpuMat > &Res,
				  ct::Size &szOut)
{
    int x1 = szOut.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	internal::SmallMtxArray sX(X), sRes(Res);

	switch (X[0].type) {
		case GPU_DOUBLE:
			internal::im2colsT_vecSame<double> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
		case GPU_FLOAT:
			internal::im2colsT_vecSame<float> <<<dimGrid, dimBlock>>>(sX, szA0, channels, szW, stride, sRes, szOut);
			break;
	}
}

////////// end same

extern "C"
void cuda_cols2im(const gpumat::GpuMat &Delta,
				const ct::Size &szOut,
				const ct::Size &szA0,
				int channels,
				const ct::Size &szW,
				int stride,
				gpumat::GpuMat &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::cols2im<double> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2im<float> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
	}
}

extern "C"
void cuda_cols2im_vec(const std::vector< gpumat::GpuMat > &Delta,
				const ct::Size &szOut,
				const ct::Size &szA0,
				int channels,
				const ct::Size &szW,
				int stride,
				std::vector< gpumat::GpuMat > &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (Delta[0].type) {
		case GPU_DOUBLE:
			internal::cols2im_vec<double> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2im_vec<float> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
	}
}

//////////////////

extern "C"
void cuda_cols2imT(const gpumat::GpuMat &Delta,
				const ct::Size &szOut,
				const ct::Size &szA0,
				int channels,
				const ct::Size &szW,
				int stride,
				gpumat::GpuMat &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::cols2imT<double> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2imT<float> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
	}
}

extern "C"
void cuda_col2imT_vec(const std::vector< gpumat::GpuMat > &Delta,
				const ct::Size &szOut,
				const ct::Size &szA0,
				int channels,
				const ct::Size &szW,
				int stride,
				std::vector< gpumat::GpuMat > &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (Delta[0].type) {
		case GPU_DOUBLE:
			internal::cols2imT_vec<double> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2imT_vec<float> <<<dimGrid, dimBlock>>>(Delta, szOut, szA0, channels, szW, stride, X);
			break;
	}
}

/////////// SAME //////////

extern "C"
void cuda_cols2im_same(const gpumat::GpuMat &Delta, const ct::Size &szDelta,
					   const ct::Size &szA0, int channels, const ct::Size &szW,
					   int stride, gpumat::GpuMat &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::cols2im_same<double> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2im_same<float> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
	}
}

extern "C"
void cuda_cols2im_vec_same(const std::vector< gpumat::GpuMat > &Delta,
						   const ct::Size &szDelta, const ct::Size &szA0,
						   int channels, const ct::Size &szW, int stride,
						   std::vector< gpumat::GpuMat > &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (Delta[0].type) {
		case GPU_DOUBLE:
			internal::cols2im_vec_same<double> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2im_vec_same<float> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
	}
}

//////////////////

extern "C"
void cuda_cols2imT_same(const gpumat::GpuMat &Delta,
						const ct::Size &szDelta, const ct::Size &szA0,
						int channels,
						const ct::Size &szW,
						int stride,
						gpumat::GpuMat &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::cols2imT_same<double> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2imT_same<float> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
	}
}

extern "C"
void cuda_col2imT_vec_same(const std::vector< gpumat::GpuMat > &Delta,
						   const ct::Size &szDelta, const ct::Size &szA0,
						   int channels,
						   const ct::Size &szW,
						   int stride,
						   std::vector< gpumat::GpuMat > &X)
{
    int x1 = szA0.area() * channels / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (Delta[0].type) {
		case GPU_DOUBLE:
			internal::cols2imT_vec_same<double> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
		case GPU_FLOAT:
			internal::cols2imT_vec_same<float> <<<dimGrid, dimBlock>>>(Delta, szDelta, szA0, channels, szW, stride, X);
			break;
	}
}

///////////////////////////

extern "C"
void cuda_subsample2(const gpumat::GpuMat &X,
							  const ct::Size &szA,
							  gpumat::GpuMat &Y,
							  gpumat::GpuMat &Mask,
							  ct::Size &szO)
{
	int K = X.cols;
    int x1 = szO.area() * K / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::subsample<double> <<<dimGrid, dimBlock>>>(X, K, szA, Y, Mask, szO);
			break;
		case GPU_FLOAT:
			internal::subsample<float> <<<dimGrid, dimBlock>>>(X, K, szA, Y, Mask, szO);
			break;
	}
}

extern "C"
void cuda_subsample2_vec(const std::vector< gpumat::GpuMat > &X,
					const ct::Size &szA,
					std::vector< gpumat::GpuMat > &Y,
					std::vector< gpumat::GpuMat > &Mask,
					ct::Size &szO)
{
	int K = X[0].cols;
    int x1 = szO.area() * K / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (X[0].type) {
		case GPU_DOUBLE:
			internal::subsample_vec<double> <<<dimGrid, dimBlock>>>(X, K, szA, Y, Mask, szO);
			break;
		case GPU_FLOAT:
			internal::subsample_vec<float> <<<dimGrid, dimBlock>>>(X, K, szA, Y, Mask, szO);
			break;
	}
}

extern "C"
void cuda_upsample2(const gpumat::GpuMat &Y, const gpumat::GpuMat &Mask, const ct::Size &szO,
			  const ct::Size &szA, gpumat::GpuMat &X)
{
	int K = X.cols;
    int x1 = szO.area() * K / BLOCKSIZE_CNV + 1;
	int x2 = 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, 1);

	switch (X.type) {
		case GPU_DOUBLE:
			internal::upsample<double> <<<dimGrid, dimBlock>>>(Y, Mask, K, szO, szA, X);
			break;
		case GPU_FLOAT:
			internal::upsample<float> <<<dimGrid, dimBlock>>>(Y, Mask, K, szO, szA, X);
			break;
	}
}

extern "C"
void cuda_upsample2vec(const std::vector<gpumat::GpuMat> &Y, const std::vector<gpumat::GpuMat> &Mask,
			  const ct::Size &szO, const ct::Size &szA, std::vector<gpumat::GpuMat> &X)
{
	int K = X[0].cols;
    int x1 = szO.area() * K / BLOCKSIZE_CNV + 1;
    int x2 = (int)X.size() / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (X[0].type) {
		case GPU_DOUBLE:
			internal::upsample_vec<double> <<<dimGrid, dimBlock>>>(Y, Mask, K, szO, szA, X);
			break;
		case GPU_FLOAT:
			internal::upsample_vec<float> <<<dimGrid, dimBlock>>>(Y, Mask, K, szO, szA, X);
			break;
	}
}


extern "C"
void cuda_vec2mat(const std::vector< GpuMat >& vec, GpuMat& mat)
{
	int rows = mat.rows;
	int cols = mat.cols;

    int x1 = cols / BLOCKSIZE_CNV + 1;
    int x2 = rows / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (vec[0].type) {
		case GPU_DOUBLE:
			internal::vec2mat<double> <<<dimGrid, dimBlock>>>(vec, mat);
			break;
		case GPU_FLOAT:
			internal::vec2mat<float> <<<dimGrid, dimBlock>>>(vec, mat);
			break;
	}
}

extern "C"
void cuda_mat2vec(const GpuMat& mat, const ct::Size& sz, std::vector< GpuMat >& vec)
{
	int rows = mat.rows;
	int cols = mat.cols;

    int x1 = cols / BLOCKSIZE_CNV + 1;
    int x2 = rows / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (vec[0].type) {
		case GPU_DOUBLE:
			internal::mat2vec<double> <<<dimGrid, dimBlock>>>(mat, sz, vec);
			break;
		case GPU_FLOAT:
			internal::mat2vec<float> <<<dimGrid, dimBlock>>>(mat, sz, vec);
			break;
	}
}

extern "C"
void cuda_addvec(gpumat::GpuMat &W, const std::vector<gpumat::GpuMat> &vW, double alpha)
{
	int rows = W.rows;
	int cols = W.cols;

    int x1 = cols / BLOCKSIZE_CNV + 1;
    int x2 = rows / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV);

	switch (W.type) {
		case GPU_DOUBLE:
			internal::addvec<double> <<<dimGrid, dimBlock>>>(W, vW, alpha);
			break;
		case GPU_FLOAT:
			internal::addvec<float> <<<dimGrid, dimBlock>>>(W, vW, alpha);
			break;
	}
}

extern "C"
void cuda_batch_normalize(_BN &bn)
{
	int spatial = bn.X->front().total() / bn.channels;

	int rows = bn.X->size();
	int cols = bn.X->front().total();

    int x1 = cols / BLOCKSIZE_CNV + 1;
    int x2 = rows / BLOCKSIZE_CNV + 1;

    dim3 dimGrid(x1, x2), dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV), dimGridC(bn.channels / BLOCKSIZE_CNV + 1, x2);

	switch (bn.X->front().type) {
		case GPU_DOUBLE:
			internal::get_mean	<double><<<dimGridC, dimBlock>>>(*bn.X, bn.Mean, spatial, bn.channels);
			internal::get_var	<double><<<dimGridC, dimBlock>>>(*bn.X, bn.Mean, bn.Xu, bn.Var, spatial, bn.channels);
			internal::get_shift	<double><<<dimGridC, dimBlock>>>(bn.Xu, bn.gamma, bn.betha, bn.Var, *bn.Y, spatial, bn.channels);
//			internal::batch_normalize<double> <<<dimGrid, dimBlock>>>(*bn.X, bn.Mean, bn.Var, bn.Xu, *bn.Y, bn.gamma, bn.betha);
			break;
		case GPU_FLOAT:
			internal::get_mean	<float ><<<dimGridC, dimBlock>>>(*bn.X, bn.Mean, spatial, bn.channels);
			internal::get_var	<float ><<<dimGridC, dimBlock>>>(*bn.X, bn.Mean, bn.Xu, bn.Var, spatial, bn.channels);
			internal::get_shift	<float ><<<dimGridC, dimBlock>>>(bn.Xu, bn.gamma, bn.betha, bn.Var, *bn.Y, spatial, bn.channels);
//			internal::batch_normalize<float> <<<dimGrid, dimBlock>>>(*bn.X, bn.Mean, bn.Var, bn.Xu, *bn.Y, bn.gamma, bn.betha);
			break;
	}
}

/////////////////////////////////

namespace gpumat{

namespace internal{

template< typename T >
__global__ void get_dbetha(const SmallMtxArray D, const Mtx gamma, Mtx dbetha, SmallMtxArray Dout, int spatial, int channels)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ T data[BLOCKSIZE_CNV][BLOCKSIZE_CNV];

	T val;
	int offr, cntr;
	int coli = threadIdx.x; int rowi = threadIdx.y;
	get_offset_and_cnt(offr, cntr, D.count, rowi);

	if(col < channels){
		T *dG	= (T*)gamma.data;
		T *ddB	= (T*)dbetha.data;

		val = 0;
		for(int i = 0; i < cntr; ++i){
			T *dD		= (T*)D.mtx[offr + i].data;
			T *dDout		= (T*)Dout.mtx[offr + i].data;
			for(int s = 0; s < spatial; ++s){
				val						   += dD[col + s * channels];
				dDout[col + s * channels]	= dD[col + s * channels] * dG[col];
			}
		}
		data[rowi][coli] = val;

		__syncthreads();

		if(rowi == 0){
			T val = 0;
			for(int i = 0; i < cntr; ++i){
				val += data[i][coli];
			}
			ddB[col] = val;
		}
	}
}

template< typename T >
__global__ void get_dgamma(const SmallMtxArray D, const SmallMtxArray Xu, const Mtx Var, Mtx dgamma, int spatial, int channels)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	T val;
	int offr, cntr;
	int coli = threadIdx.x; int rowi = threadIdx.y;
	get_offset_and_cnt(offr, cntr, D.count, rowi);
    __shared__ T data[BLOCKSIZE_CNV][BLOCKSIZE_CNV];

	if(col < channels){
		T *dG	= (T*)dgamma.data;
		T *dV = (T*)Var.data;

		val = 0;
		for(int i = 0; i < cntr; ++i){
			T *dDi		= (T*)D.mtx[offr + i].data;
			T *dXu		= (T*)Xu.mtx[offr + i].data;
			for(int s = 0; s < spatial; ++s){
				val			+= dDi[col + s * channels] * (dXu[col + s * channels] / dV[col]);
			}
		}
		data[rowi][coli] = val;

		__syncthreads();

		if(rowi == 0){
			T val = 0;
			for(int i = 0; i < cntr; ++i){
				val += data[i][coli];
			}
			dG[col] = val;
		}
	}
}

template< typename T >
__global__ void get_dxmu(const SmallMtxArray D, const Mtx Var, SmallMtxArray Dout, int spatial, int channels)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if(col < channels && row < Dout.count){
		T *dV	= (T*)Var.data;

		T *dD		= (T*)D.mtx[row].data;
		T *dDout	= (T*)Dout.mtx[row].data;
		for(int s = 0; s < spatial; ++s){
			dDout[col + s * channels]	= dD[col + s * channels] / dV[col];
		}
	}
}

template< typename T >
__global__ void get_dsigma(const SmallMtxArray D, const SmallMtxArray Xmu, const Mtx Var, Mtx DVar, int spatial, int channels)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	T N = D.count * spatial, val;
	int offr, cntr;
	int coli = threadIdx.x; int rowi = threadIdx.y;
	get_offset_and_cnt(offr, cntr, D.count, rowi);
    __shared__ T data[BLOCKSIZE_CNV][BLOCKSIZE_CNV];

	if(col < channels){
		T *dV	= (T*)Var.data;

		val = 0;
		for(int i = 0; i < cntr; ++i){
			T *dD		= (T*)D.mtx[offr + i].data;
			T *dXmu		= (T*)Xmu.mtx[offr + i].data;
			for(int s = 0; s < spatial; ++s){
				val += dD[col + s * channels] * dXmu[col + s * channels];
			}
		}
		data[rowi][coli] = val;

		__syncthreads();

		if(rowi == 0){
			T *ddV	= (T*)DVar.data;

			T val = 0;
			for(int i = 0; i < cntr; ++i){
				val += data[i][coli];
			}
			T s = dV[col];
			T res = -val * (1/(s * s));
			res = res * 0.5 * (1/s);
			ddV[col] = res / (N - 1);
		}
	}
}

template< typename T >
__global__ void get_dmean(const SmallMtxArray D, const SmallMtxArray Xmu, const Mtx DVar, Mtx dMean, SmallMtxArray Dout, int spatial, int channels)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	T N = Dout.count * spatial, val;
	int offr, cntr;
	int coli = threadIdx.x; int rowi = threadIdx.y;
	get_offset_and_cnt(offr, cntr, Dout.count, rowi);
    __shared__ T data[BLOCKSIZE_CNV][BLOCKSIZE_CNV];

	if(col < channels){
		T *dA	= (T*)dMean.data;
		T *dDVar = (T*)DVar.data;

		val = 0;
		for(int i = 0; i < cntr; ++i){
			T *dD			= (T*)D.mtx[offr + i].data;
			T *dDout		= (T*)Dout.mtx[offr + i].data;
			T *dXmu			= (T*)Xmu.mtx[offr + i].data;
			for(int s = 0; s < spatial; ++s){
				T dsq						= 2 * dDVar[col] * dXmu[col + s * channels];
				val							-= (dsq + dD[col + s * channels]);
				dDout[col + s * channels]	= dD[col + s * channels] + dsq;
			}
		}
		data[rowi][coli] = val;

		__syncthreads();

		if(rowi == 0){
			T val = 0;
			for(int i = 0; i < cntr; ++i){
				val += data[i][coli];
			}
			dA[col] = val/N;
		}
	}
}

template< typename T >
__global__ void add2mean(const SmallMtxArray Dout, const Mtx DMean, SmallMtxArray Xout, int spatial, int channels)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if(col < channels && row < Dout.count){
		T *dDMean	= (T*)DMean.data;

		T *dDout	= (T*)Dout.mtx[row].data;
		T *dXout	= (T*)Xout.mtx[row].data;
		for(int s = 0; s < spatial; ++s){
			T xout = dDout[col + s * channels] + dDMean[col];
			dXout[col + s * channels] = xout;
		}
	}
}
template< typename T >
__global__ void scale_and_shift(const SmallMtxArray X, const Mtx gamma, Mtx betha, SmallMtxArray Y, int spatial, int channels)
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if(col < channels && row < X.count){
		T *dG	= (T*)gamma.data;
		T *dB	= (T*)betha.data;

		T *dX	= (T*)X.mtx[row].data;
		T *dY	= (T*)Y.mtx[row].data;
		for(int s = 0; s < spatial; ++s){
			T xout = dX[col + s * channels];
			dY[col + s * channels] = dG[col] * xout + dB[col];
		}
	}
}

}	/** @endnamespace internal */

}	/** @endnamespace gpumat */

extern "C"
void cuda_batch_denormalize(_BN &bn)
{
	int spatial = bn.X->front().total() / bn.channels;

	int rows = bn.X->size();
	int cols = bn.X->front().total();

//	int x1 = cols / BLOCKSIZE_CNV + 1;
    int x2 = rows / BLOCKSIZE_CNV + 1;

    dim3 dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV), dimGridC(bn.channels / BLOCKSIZE_CNV + 1, x2);

	bn.dgamma.resize(bn.gamma);
	bn.dbetha.resize(bn.betha);
//	bn.dMean.resize(bn.Mean);
	bn.dVar.resize(bn.Var);

	switch (bn.D->front().type) {
		case GPU_DOUBLE:
			internal::get_dbetha	<double> <<<dimGridC, dimBlock>>>(*bn.D, bn.gamma, bn.dbetha, bn.Dout, spatial, bn.channels);
			internal::get_dgamma	<double> <<<dimGridC, dimBlock>>>(*bn.D, bn.Xu, bn.Var, bn.dgamma, spatial, bn.channels);
			internal::get_dxmu		<double> <<<dimGridC, dimBlock>>>(bn.Dout, bn.Var, *bn.D, spatial, bn.channels);
			internal::get_dsigma	<double> <<<dimGridC, dimBlock>>>(bn.Dout, bn.Xu, bn.Var, bn.dVar, spatial, bn.channels);
			internal::get_dmean		<double> <<<dimGridC, dimBlock>>>(*bn.D, bn.Xu, bn.dVar, bn.Mean, bn.Dout, spatial, bn.channels);
			internal::add2mean		<double> <<<dimGridC, dimBlock>>>(bn.Dout, bn.Mean, bn.Dout, spatial, bn.channels);
			break;
		case GPU_FLOAT:
			internal::get_dbetha	<float > <<<dimGridC, dimBlock>>>(*bn.D, bn.gamma, bn.dbetha, bn.Dout, spatial, bn.channels);
			internal::get_dgamma	<float > <<<dimGridC, dimBlock>>>(*bn.D, bn.Xu, bn.Var, bn.dgamma, spatial, bn.channels);
			internal::get_dxmu		<float > <<<dimGridC, dimBlock>>>(bn.Dout, bn.Var, *bn.D, spatial, bn.channels);
			internal::get_dsigma	<float > <<<dimGridC, dimBlock>>>(bn.Dout, bn.Xu, bn.Var, bn.dVar, spatial, bn.channels);
			internal::get_dmean		<float > <<<dimGridC, dimBlock>>>(*bn.D, bn.Xu, bn.dVar, bn.Mean, bn.Dout, spatial, bn.channels);
			internal::add2mean		<float > <<<dimGridC, dimBlock>>>(bn.Dout, bn.Mean, bn.Dout, spatial, bn.channels);
			break;
	}
//	int index = 0;
//	for(GpuMat &D: *bn.D){
//		D.copyTo(bn.Dout[index++]);
//	}
}

extern "C"
void cuda_scale_and_shift_bn(_BN &bn)
{
	int spatial = bn.X->front().total() / bn.channels;

	int rows = bn.X->size();
	int cols = bn.X->front().total();

//	int x1 = cols / BLOCKSIZE_CNV + 1;
    int x2 = rows / BLOCKSIZE_CNV + 1;

    dim3 dimBlock(BLOCKSIZE_CNV, BLOCKSIZE_CNV), dimGridC(bn.channels / BLOCKSIZE_CNV + 1, x2);

	switch (bn.D->front().type) {
		case GPU_DOUBLE:
			internal::scale_and_shift	<double> <<<dimGridC, dimBlock>>>(*bn.X, bn.gamma, bn.betha, *bn.Y, spatial, bn.channels);
			break;
		case GPU_FLOAT:
			internal::scale_and_shift	<float > <<<dimGridC, dimBlock>>>(*bn.X, bn.gamma, bn.betha, *bn.Y, spatial, bn.channels);
			break;
	}

}
